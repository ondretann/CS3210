#include "hip/hip_runtime.h"
#include "kseq/kseq.h"
#include "common.h"
#include <iostream>

struct KSeqChar {
    const char* seq;
    const char* qual;
    int len;
};

__device__ double calc_match_confidence(const KSeqChar* sample, int start, int sig_len) {
    double score = 0.0;
    const char* qual = sample->qual;
    for (int i = 0; i < sig_len; i++) {
        score += qual[start + i] - 33;
    }
    return score / sig_len;
}

__device__ double sample_sign_match(const KSeqChar* sample, const KSeqChar* signature) {
    int samp_len = sample->len;
    int sig_len = signature->len;
    const char* samp_seq = sample->seq;
    const char* sig_seq = signature->seq;

    if (samp_len >= sig_len) {
        // Sliding window over the sample sequence
        for (int i = 0; i <= samp_len - sig_len; i++) {
            bool match = true;
            for (int j = 0; j < sig_len; j++) {
                if (samp_seq[i + j] == 'N' || sig_seq[j] == 'N') {
                    continue;
                }

                if (samp_seq[i + j] != sig_seq[j]) {
                    match = false;
                    break;
                }
            }
            if (match) {
                return calc_match_confidence(sample, i, sig_len);
            }
        }
    }
    return -1.0; 
}

__global__ void matcher_kernel(KSeqChar* samples, KSeqChar* signatures, double* con_scores, int num_samples, int num_signatures) {
    int sample_idx = blockIdx.x;
    int signature_idx = threadIdx.x;

    double match_score = sample_sign_match(&samples[sample_idx], &signatures[signature_idx]);

    // Write the result to the appropriate position in the result array
    if (match_score >= 0.0) {
        int result_index = sample_idx * num_signatures + signature_idx;
        con_scores[result_index] = match_score;
    }
}

void runMatcher(const std::vector<klibpp::KSeq>& samples, const std::vector<klibpp::KSeq>& signatures, std::vector<MatchResult>& matches) {
    // preprocessing begin
    int num_samples = samples.size();
    int num_signatures = signatures.size();

    KSeqChar* kseq_sample = new KSeqChar[num_samples];
    KSeqChar* kseq_sign = new KSeqChar[num_signatures];

    for (int i = 0; i < num_samples; i++) {
        kseq_sample[i].len = samples[i].seq.size();

        char* d_seq;
        hipMalloc((void**)&d_seq, samples[i].seq.size() + 1);
        hipMemcpy(d_seq, samples[i].seq.c_str(), samples[i].seq.size() + 1, hipMemcpyHostToDevice);
        kseq_sample[i].seq = d_seq;

        char* d_qual;
        hipMalloc((void**)&d_qual, samples[i].qual.size() + 1);
        hipMemcpy(d_qual, samples[i].qual.c_str(), samples[i].qual.size() + 1, hipMemcpyHostToDevice);
        kseq_sample[i].qual = d_qual;
    }

    for (int i = 0; i < num_signatures; i++) {
        kseq_sign[i].len = signatures[i].seq.size();

        char* d_seq;
        hipMalloc((void**)&d_seq, signatures[i].seq.size() + 1);
        hipMemcpy(d_seq, signatures[i].seq.c_str(), signatures[i].seq.size() + 1, hipMemcpyHostToDevice);
        kseq_sign[i].seq = d_seq;
    }

    KSeqChar* d_samples;
    KSeqChar* d_signatures;
    hipMalloc(&d_samples, num_samples * sizeof(KSeqChar));
    hipMalloc(&d_signatures, num_signatures * sizeof(KSeqChar));

    hipMemcpy(d_samples, kseq_sample, num_samples * sizeof(KSeqChar), hipMemcpyHostToDevice);
    hipMemcpy(d_signatures, kseq_sign, num_signatures * sizeof(KSeqChar), hipMemcpyHostToDevice);

    int array_size = num_samples * num_signatures;
    double* d_con_scores;
    hipMalloc(&d_con_scores, array_size * sizeof(double));
    hipMemset(d_con_scores, -1, array_size * sizeof(double));

    dim3 gridDim(num_samples);
    dim3 blockDim(num_signatures);

    matcher_kernel<<<gridDim, blockDim>>>(d_samples, d_signatures, d_con_scores, num_samples, num_signatures);
    hipDeviceSynchronize();
    
    double* h_con_scores = new double[array_size];
    hipMemcpy(h_con_scores, d_con_scores, array_size * sizeof(double), hipMemcpyDeviceToHost);

    for (int k = 0; k < array_size; k++) {
        if (h_con_scores[k] >= 0.0) {
            MatchResult result;
            int samp_index = k / num_signatures;
            int sign_index = k % num_signatures;
            result.sample_name = samples[samp_index].name;
            result.signature_name = signatures[sign_index].name;
            result.match_score = h_con_scores[k];
            matches.push_back(result);
        }
    }

    delete[] kseq_sample;
    delete[] kseq_sign;
    delete[] h_con_scores;

    hipFree(d_samples);
    hipFree(d_signatures);
    hipFree(d_con_scores);
}
